// scalar_add_inplace.cu
#include <hip/hip_runtime.h>

__global__ void scalarAddKernel(float *input, float *output, int rows, int cols, float scalar) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows) {
        int index = idy * cols + idx;
        output[index] = input[index] + scalar;
    }
}
