// transpose.cu
#include <hip/hip_runtime.h>

__global__ void transposeKernel(float *input, float *output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows) {
        output[idx * rows + idy] = input[idy * cols + idx];
    }
}